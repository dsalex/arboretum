#include <stdio.h>
#include <limits>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/system/cuda/execution_policy.h>
#include "garden.h"
#include "param.h"
#include "objective.h"


namespace arboretum {
  namespace core {
    using namespace std;
    using namespace thrust;
    using namespace thrust::cuda;
    using thrust::host_vector;
    using thrust::device_vector;


    template <typename T>
    struct max_gain_functor{
      typedef T first_argument_type;
      typedef T second_argument_type;
      typedef T result_type;

      max_gain_functor(){}

      __host__ __device__
      T operator()(const T &l, const T &r) const {
        const T _lookup [2] { r, l };
        return _lookup[thrust::get<0>(l) > thrust::get<0>(r)];
      }
    };

    struct gain_functor{
      const int min_wieght;

      gain_functor(int min_wieght) : min_wieght(min_wieght) {}

      template <typename Tuple>
      __host__ __device__
      void operator()(Tuple t)
      {
        const double left_sum = thrust::get<0>(t);
        const size_t left_count = thrust::get<1>(t);
        const double total_sum = thrust::get<2>(t);
        const size_t total_count = thrust::get<3>(t);
        const float fvalue = thrust::get<5>(t);
        const float fvalue_prev = thrust::get<6>(t);
        const size_t right_count = total_count - left_count;

        if(left_count >= min_wieght && right_count >= min_wieght && fvalue != fvalue_prev){
            const size_t d = left_count * total_count * (total_count - left_count);

            const double top = total_count * left_sum - left_count * total_sum;
            thrust::get<4>(t) = top*top/d;
          } else {
            thrust::get<4>(t) = 0.0;
          }
      }
    };

    class GardenBuilder : public GardenBuilderBase {
    public:
      GardenBuilder(const TreeParam &param, const io::DataMatrix* data) : param(param){
        _rowIndex2Node.resize(data->rows, 0);
        _featureNodeSplitStat.resize(data->columns);
        _bestSplit.resize(1 << (param.depth - 2));
        _nodeStat.resize(1 << (param.depth - 2));
        for(size_t fid = 0; fid < data->columns; ++fid){
            _featureNodeSplitStat[fid].resize(1 << param.depth);
          }
    }
      virtual void InitGrowingTree() override {
        std::fill(_rowIndex2Node.begin(), _rowIndex2Node.end(), 0);
        for(size_t i = 0; i < _featureNodeSplitStat.size(); ++i){
            for(size_t j = 0; j < _featureNodeSplitStat[i].size(); ++j){
                _featureNodeSplitStat[i][j].Clean();
              }
          }
        for(size_t i = 0; i < _nodeStat.size(); ++i){
            _nodeStat[i].Clean();
          }
        for(size_t i = 0; i < _bestSplit.size(); ++i){
            _bestSplit[i].Clean();
          }
      }

      virtual void InitTreeLevel(const int level) override {
        for(size_t i = 0; i < _featureNodeSplitStat.size(); ++i){
            for(size_t j = 0; j < _featureNodeSplitStat[i].size(); ++j){
                _featureNodeSplitStat[i][j].Clean();
              }
          }
      }

      virtual void GrowTree(RegTree *tree, const io::DataMatrix *data, const thrust::host_vector<float> &grad) override{
        InitGrowingTree();

        for(int i = 0; i < param.depth - 1; ++i){
          InitTreeLevel(i);
          UpdateNodeStat(i, grad, tree);
          FindBestSplits(i, data, grad);
          UpdateTree(i, tree);
          UpdateNodeIndex(i, data, tree);
        }

        UpdateLeafWeight(tree);
      }

      virtual void PredictByGrownTree(RegTree *tree, const io::DataMatrix *data, std::vector<float> &out) override {
        tree->Predict(data, _rowIndex2Node, out);
      }

    private:
      const TreeParam param;
      host_vector<unsigned int> _rowIndex2Node;
      std::vector<std::vector<SplitStat> > _featureNodeSplitStat;
      std::vector<NodeStat> _nodeStat;
      std::vector<Split> _bestSplit;

      void FindBestSplits(const int level, const io::DataMatrix *data, const thrust::host_vector<float> &grad){

                      device_vector<unsigned int> row2Node = _rowIndex2Node;
                      device_vector<unsigned int> segments(data->rows);
                      device_vector<float> grad_sorted(data->rows);
                      device_vector<float> fvalue(data->rows + 1);
                      fvalue[0] = -std::numeric_limits<float>::infinity();

                      for(size_t fid = 0; fid < data->columns; ++fid){

                          thrust::copy(data->sorted_data[fid].begin(), data->sorted_data[fid].end(), fvalue.begin() + 1);
                          device_vector<int> position(data->index[fid].begin(), data->index[fid].end());

                          thrust::gather(position.begin(),
                                         position.end(),
                                         row2Node.begin(),
                                         segments.begin());

                          thrust::gather(position.begin(),
                                         position.end(),
                                         data->grad_device.begin(),
                                         grad_sorted.begin());

                          thrust::stable_sort_by_key(segments.begin(),
                                                     segments.end(),
                                                     thrust::make_zip_iterator(
                                                       thrust::make_tuple(grad_sorted.begin(),
                                                       fvalue.begin() + 1)
                                                       ));

                          device_vector<double> sum(data->rows);
                          device_vector<double> gain(data->rows);
                          device_vector<size_t> count(data->rows, 1);

                          thrust::equal_to<unsigned int> binary_pred;

                          thrust::exclusive_scan_by_key(segments.begin(),
                                                        segments.end(),
                                                        grad_sorted.begin(),
                                                        sum.begin());

                          thrust::exclusive_scan_by_key(segments.begin(),
                                                        segments.end(),
                                                        count.begin(),
                                                        count.begin());

                          size_t lenght = 1 << level;

                          device_vector<double> parent_node_sum(lenght);
                          device_vector<size_t> parent_node_count(lenght);

                          for(size_t i = 0; i < lenght; ++i){
                              parent_node_count[i] = _nodeStat[i].count;
                              parent_node_sum[i] = _nodeStat[i].sum_grad;
                            }

                          device_vector<double> parent_node_sum_vector(data->rows, 0.0);
                          device_vector<size_t> parent_node_count_vector(data->rows, 0);

                          thrust::gather(segments.begin(),
                                         segments.end(),
                                         parent_node_sum.begin(),
                                         parent_node_sum_vector.begin());

                          thrust::gather(segments.begin(),
                                         segments.end(),
                                         parent_node_count.begin(),
                                         parent_node_count_vector.begin());

                          thrust::for_each(
                                thrust::make_zip_iterator(
                                  thrust::make_tuple(sum.begin(), count.begin(), parent_node_sum_vector.begin(),
                                                     parent_node_count_vector.begin(), gain.begin(),
                                                     fvalue.begin() + 1, fvalue.begin())),
                                thrust::make_zip_iterator(
                                  thrust::make_tuple(sum.end(), count.end(), parent_node_sum_vector.end(),
                                                     parent_node_count_vector.end(), gain.end(),
                                                     fvalue.end(), fvalue.end() - 1)),
                              gain_functor(param.min_child_weight));

                          device_vector<int> max_key_d(1 << level, -1);
                          device_vector<thrust::tuple<double, float, float, size_t, double>> max_value_d(1 << level);

                          host_vector<int> max_key(1 << level, -1);
                          host_vector<thrust::tuple<double, float, float, size_t, double>> max_value(1 << level);


                          auto tuple_iterator = thrust::make_zip_iterator(
                                thrust::make_tuple(gain.begin(),
                                                   fvalue.begin(),
                                                   fvalue.begin() + 1,
                                                   count.begin(),
                                                   sum.begin()));

                          max_gain_functor< thrust::tuple<double, float, float, size_t, double> > binary_op;

                          thrust::reduce_by_key(segments.begin(),
                                                segments.end(),
                                                tuple_iterator,
                                                max_key_d.begin(),
                                                max_value_d.begin(),
                                                binary_pred,
                                                binary_op);

                          thrust::copy(max_key_d.begin(), max_key_d.end(), max_key.begin());
                          thrust::copy(max_value_d.begin(), max_value_d.end(), max_value.begin());

                          for(size_t i = 0; i < max_key.size(); ++i){
                              const int node_index = max_key[i];
                              const thrust::tuple<double, float, float, size_t, double> t = max_value[i];
                              const double gain_value = thrust::get<0>(t);
                              const float fvalue_prev_val = thrust::get<1>(t);
                              const float fvalue_val = thrust::get<2>(t);
                              const size_t count_val = thrust::get<3>(t);
                              const double sum_val = thrust::get<4>(t);

                              if(node_index >= 0){
                                  if(gain_value > _bestSplit[node_index].gain){
                                      _bestSplit[node_index].fid = fid;
                                      _bestSplit[node_index].gain = gain_value;
                                      _bestSplit[node_index].split_value = (fvalue_prev_val + fvalue_val) * 0.5;
                                      _bestSplit[node_index].count = count_val;
                                      _bestSplit[node_index].sum_grad = sum_val;
                                    }
                                }
                            }

                          for(size_t i = 0; i < lenght; ++i){
                              NodeStat &node_stat = _nodeStat[i];
                              Split &split = _bestSplit[i];

                              if(split.fid < 0){
                                  _bestSplit[i].gain = 0.0;
                                  _bestSplit[i].fid = 0;
                                  _bestSplit[i].split_value = std::numeric_limits<float>::infinity();
                                  _bestSplit[i].count = node_stat.count;
                                  _bestSplit[i].sum_grad = node_stat.sum_grad;
                                }
                            }

                        }

      }
      void UpdateNodeStat(const int level, const thrust::host_vector<float> &grad, const RegTree *tree){
        if(level != 0){
        const unsigned int offset = Node::HeapOffset(level);
        const unsigned int offset_next = Node::HeapOffset(level + 1);
        std::vector<NodeStat> tmp(_nodeStat.size());
        std::copy(_nodeStat.begin(), _nodeStat.end(), tmp.begin());
        for(size_t i = 0, len = 1 << (level - 1); i < len; ++i){
            _nodeStat[tree->ChildNode(i + offset, true) - offset_next].count = _bestSplit[i].count;
            _nodeStat[tree->ChildNode(i + offset, true) - offset_next].sum_grad = _bestSplit[i].sum_grad;

            _nodeStat[tree->ChildNode(i + offset, false) - offset_next].count =
                tmp[i].count - _bestSplit[i].count;

            _nodeStat[tree->ChildNode(i + offset, false) - offset_next].sum_grad =
                tmp[i].sum_grad - _bestSplit[i].sum_grad;

            _bestSplit[i].Clean();
          }
          } else {
            for(size_t i = 0; i < grad.size(); ++i){
                int node = _rowIndex2Node[i];
                _nodeStat[node].count++;
                _nodeStat[node].sum_grad += grad[i];
              }
          }
        for(size_t i = 0, len = 1 << level; i < len; ++i){
            _nodeStat[i].gain = (_nodeStat[i].sum_grad * _nodeStat[i].sum_grad) / _nodeStat[i].count;
            _bestSplit[i].Clean();
          }
      }

      void UpdateTree(const int level, RegTree *tree) const {
        unsigned int offset = Node::HeapOffset(level);
        for(size_t i = 0, len = 1 << level; i < len; ++i){
            const Split &best = _bestSplit[i];
            tree->nodes[i + offset].threshold = best.split_value;
            tree->nodes[i + offset].fid = best.fid;
            if(tree->nodes[i + offset].fid < 0){
                tree->nodes[i + offset].fid = 0;
              }
          }
      }

      void UpdateNodeIndex(const unsigned int level, const io::DataMatrix *data, RegTree *tree) {
        unsigned int offset = Node::HeapOffset(level);
        unsigned int offset_next = Node::HeapOffset(level + 1);
        unsigned int node;
        for(size_t i = 0; i < data->rows; ++i){
            node = _rowIndex2Node[i];
            Split &best = _bestSplit[node];
            _rowIndex2Node[i] = tree->ChildNode(node + offset, data->data[best.fid][i] <= best.split_value) - offset_next;
          }
      }

      void UpdateLeafWeight(RegTree *tree) const {
        const unsigned int offset_1 = Node::HeapOffset(tree->depth - 2);
        const unsigned int offset = Node::HeapOffset(tree->depth - 1);
        for(unsigned int i = 0, len = (1 << (tree->depth - 2)); i < len; ++i){
            const Split &best = _bestSplit[i];
            const NodeStat &stat = _nodeStat[i];
            tree->leaf_level[tree->ChildNode(i + offset_1, true) - offset] = (best.sum_grad / best.count) * param.eta * (-1);
            tree->leaf_level[tree->ChildNode(i + offset_1, false) - offset] = ((stat.sum_grad - best.sum_grad) / (stat.count - best.count)) * param.eta * (-1);
          }
      }
    };

    Garden::Garden(const TreeParam& param) : param(param), _init(false) {}
    void Garden::GrowTree(io::DataMatrix* data, float *grad){

      if(!_init){
          std::function<float const(float const, float const)> gradFunc;
          switch (param.objective) {
            case LinearRegression:
              gradFunc = GradBuilder::Regression;
              break;
            case LogisticRegression:
              gradFunc = GradBuilder::LogReg;
              break;
            default:
               throw "Unknown objective function";
              break;
            }

          data->Init(param.initial_y, gradFunc);
          _builder = new GardenBuilder(param, data);
          _init = true;
        }

      _builder->InitGrowingTree();

      if(grad == NULL){
          SetInitial(data, data->y);
          data->UpdateGrad();
        } else {
          data->grad = std::vector<float>(grad, grad + data->rows);
        }

        RegTree *tree = new RegTree(param.depth);
        _builder->GrowTree(tree, data, data->grad);
        _trees.push_back(tree);
        if(grad == NULL){
            _builder->PredictByGrownTree(tree, data, data->y);
          }
      }

    void Garden::Predict(const arboretum::io::DataMatrix *data, std::vector<float> &out){
      out.resize(data->rows);
      std::fill(out.begin(), out.end(), param.initial_y);
      for(size_t i = 0; i < _trees.size(); ++i){
          _trees[i]->Predict(data, out);
        }
    }

    void Garden::SetInitial(const arboretum::io::DataMatrix *data, std::vector<float> &out){
      if(out.size() != data->rows){
          out.resize(data->rows);
          std::fill(out.begin(), out.end(), param.initial_y);
        }
    }
    }
  }

